#include <stdio.h>
#include "GPU_Manage.h"
#include "Types.h"
#include "Constants.h"
#include "GPU_Test.h"
#include "GPU_MBR.h"
#include "IO.h"
#include "SEQ_Overlay.h"
#include "GPU_Utility.h"
#include "Data_Visualization.h"
#include "Join.h"

hipEvent_t start_GPU, stop_GPU;

int main(int argc, char* argv[]){  
    float Join_Total_Time_SEQ=0, Join_Total_Time_GPU=0;
    hipError_t cudaMemError;
//------------------------ Console Input ---------------------------------- 
/*
First user input: dimSort
	1: Sorting just based on one dimension (default is X)
	0: Sorting based on both X and Y dimensions
Second user input: dimSelect
	If dimSort=1, this argument define which dimension should be picked for sorting (Values could be 'X' or 'Y')
*/
    int dimSort=1, dimSelect=1;
    if(argc<2){
       dimSort=1;
       dimSelect=0;
    }
    else if(argc<3){
      if(argv[1][0]=='2')dimSort=2;
      else dimSort=1;
      dimSelect=0;
    }
    else if(argc<4){
      if(argv[2][0]=='y')dimSelect=1;
      else dimSelect=0;
      if(argv[1][0]=='2'){dimSort=2;dimSelect=0;}
      else dimSort=1;
    }
//------------------------------------------------------------------------------


//-------------------------------------------- Reading Input -----------------------------------------------    
    char baseFileName[100], overlayFileName[100];
    long bPolNum, oPolNum;
    switch(DATASET){
       case 1:
         bPolNum = 4646;
	 oPolNum = 11878;
         //strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/admin_states.txt");
         strcpy(baseFileName, "../admin_states.txt");
	 //strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/urban_areas.txt");
	 strcpy(overlayFileName, "../urban_areas.txt");
         printf("\nDataset: admin - urban\n");
         break;
       case 2:
         bPolNum = 15000;
	 oPolNum = 15000;
         strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/bases_242.txt");
         strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/overlay_300.txt");
         printf("\nDataset: bases - overlay\n");
         break;
       case 3:
         bPolNum = 15000;
	 oPolNum = 15000;
         strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/block_boundaries.txt");
         strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/water_bodies.txt");
         printf("\nDataset: boundaries - water\n");
         break;
       case 4:
         bPolNum = 15000;
	 oPolNum = 15000;
         strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/postal.txt");
         strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/sports.txt");
         printf("\nDataset: postal - sports\n");
         break;
    }
    //----------------------------------------- Memory Allocation -------------------------------------    
    long bVNumSum = 0, oVNumSum = 0;    
    int *bVNum=(int*)malloc(sizeof(int) * bPolNum);
    int *oVNum=(int*)malloc(sizeof(int) * oPolNum);
    long *bVPSNum=(long*)malloc(sizeof(long) * bPolNum);
    long *oVPSNum=(long*)malloc(sizeof(long) * oPolNum);

    coord_t* baseXCoords = (coord_t*) malloc( 2 * sizeof(coord_t) * (VERTEX_PER_BPOL * bPolNum + VERTEX_PER_OPOL * oPolNum) );
    coord_t* overlayXCoords = baseXCoords + VERTEX_PER_BPOL * bPolNum;
    coord_t* baseYCoords = overlayXCoords + VERTEX_PER_OPOL * oPolNum;
    coord_t* overlayYCoords = baseYCoords + VERTEX_PER_BPOL * bPolNum;

    mbr_t* seqXMBR, *seqOXMBR, *seqYMBR, *seqOYMBR;
    seqXMBR = (mbr_t*)malloc(4 * sizeof(mbr_t) * (bPolNum + oPolNum));
    seqOXMBR = seqXMBR + 2 * bPolNum;
    seqYMBR = seqOXMBR + 2 * oPolNum;
    seqOYMBR = seqYMBR + 2 * bPolNum;

    coord_t* seqXMBR2, *seqYMBR2, *seqOXMBR2, *seqOYMBR2;
    seqXMBR2 = (coord_t*)malloc(4 * sizeof(coord_t) * (bPolNum + oPolNum));
    seqOXMBR2= seqXMBR2 + 2 * bPolNum;
    seqYMBR2 = seqOXMBR2 + 2 * oPolNum;
    seqOYMBR2 = seqYMBR2 + 2 * bPolNum;

    //-------------------------------------------------------------------------------------------------    
   
    bPolNum=ReadTextFormatPolygon2(baseFileName,bVNum, bVPSNum, seqXMBR, seqYMBR, seqXMBR2, seqYMBR2, baseXCoords, baseYCoords, &bVNumSum, 1, bPolNum);    
    printf("\n%lu Polygons with %lu vertices in total.\n",bPolNum,bVNumSum);
    oPolNum=ReadTextFormatPolygon2(overlayFileName,oVNum, oVPSNum, seqOXMBR, seqOYMBR, seqOXMBR2, seqOYMBR2, overlayXCoords, overlayYCoords, &oVNumSum, 1, oPolNum);    
    printf("\n%lu Polygons with %lu vertices in total.\n",oPolNum,oVNumSum);
//-----------------------------------------------------------------------------------------------------    


//----------------------------------- Reseting GPU Device --------------------------------------------- 
    hipError_t error_reset=hipDeviceReset();    
    if(error_reset!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error_reset) );
       exit(-1);
    }
    hipDeviceReset();
    size_t mem_free_0,mem_total_0;
    hipMemGetInfo  (&mem_free_0, &mem_total_0);
    printf("\nFree: %lu  , Total: %lu\n",mem_free_0,mem_total_0);
//-----------------------------------------------------------------------------------------------------    
    
//----------------------------------- Transfering data to GPU -----------------------------------------
    StartTimer(&start_GPU, &stop_GPU);
   
    int *dbVNum, *doVNum;
    coord_t *oXCoords, *oYCoords, *bXCoords, *bYCoords;
    mbr_t *doXMBR, *doYMBR, *dbXMBR, *dbYMBR;
    long *dbVPSNum, *doVPSNum;

    //----------- Transfering polygon number variables to GPU ---------------
    CopyToGPU((void**)&dbVNum, bVNum, sizeof(int)*bPolNum, "dbVNum", 1);
    CopyToGPU((void**)&doVNum, oVNum, sizeof(int)*oPolNum, "doVNum", 1);
    CopyToGPU((void**)&dbVPSNum, bVPSNum, sizeof(long)*bPolNum, "dbVPSNum", 1);
    CopyToGPU((void**)&doVPSNum, oVPSNum, sizeof(long)*oPolNum, "doVPSNum", 1);
    //-----------------------------------------------------------------------

    //----------------------- Transfering MBRs to GPU -----------------------
    hipError_t memAlloc = hipMalloc( (void**)&dbXMBR, 4 * sizeof(mbr_t) * (bPolNum + oPolNum) ); 
    if(memAlloc != hipSuccess){printf("\nError in device memory allocation!\n");return(0);}

    CopyToGPU((void**)&dbXMBR, seqXMBR, 2 * sizeof(mbr_t) * bPolNum, "dbXMBR", 0);
    doXMBR = dbXMBR + 2 * bPolNum;
    CopyToGPU((void**)&doXMBR, seqOXMBR, 2 * sizeof(mbr_t) * oPolNum, "doXMBR", 0);
    dbYMBR = doXMBR + 2 * oPolNum;
    CopyToGPU((void**)&dbYMBR, seqYMBR, 2 * sizeof(mbr_t) * bPolNum, "dbYMBR", 0);
    doYMBR = dbYMBR + 2 * bPolNum;
    CopyToGPU((void**)&doYMBR, seqOYMBR, 2 * sizeof(mbr_t) * oPolNum, "doYMBR", 0);
    //-----------------------------------------------------------------------
    //------------- Transfering polygon coordinates to GPU ------------------
    memAlloc = hipMalloc( (void**)&bXCoords, 2 * sizeof(coord_t) * (bVNumSum + oVNumSum)); 
    if(memAlloc != hipSuccess){printf("\nError in device memory allocation!\n");return(0);}

    CopyToGPU((void**)&bXCoords, baseXCoords, sizeof(coord_t) * bVNumSum , "bXCoords", 0);
    oXCoords = bXCoords + bVNumSum;
    CopyToGPU((void**)&oXCoords, overlayXCoords, sizeof(coord_t) * oVNumSum, "oXCoords", 0);
    bYCoords = oXCoords + oVNumSum;
    CopyToGPU((void**)&bYCoords, baseYCoords, sizeof(coord_t) * bVNumSum , "bYCoords", 0);
    oYCoords = bYCoords + bVNumSum;
    CopyToGPU((void**)&oYCoords, overlayYCoords, sizeof(coord_t) * oVNumSum , "oYCoords", 0);
    //-----------------------------------------------------------------------

    GPUSync("Transfering data to GPU");

    float runningTime_GPU_TransferData;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_TransferData);
    printf("\n\nGPU running time for transfering data to GPU: %f (%f)\n",runningTime_GPU_TransferData, Join_Total_Time_GPU);
//-----------------------------------------------------------------------------------------------------    


//--------------------------- Find Overlaping MBRs (novel approach) -----------------------------------
    StartTimer(&start_GPU, &stop_GPU);

    int *djxyCounter, *djxyVector, polNum=bPolNum+oPolNum; 
    cudaMemError=hipMalloc((void**)&djxyCounter,sizeof(int)*(polNum));

    long pairNum=SortBaseMBROverlap(bPolNum, oPolNum, dbXMBR, dbYMBR, doXMBR, doYMBR, &djxyCounter, &djxyVector, dimSort, dimSelect);
   
    printf("\n\n\tPolygon pairs candidate: %ld\n", pairNum);
    float runningTime_GPU_overlap2;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_overlap2);
    printf("\nGPU Running Time For Computing MBR intersection (new approach %dD [dim:%c] ): %f (%f)\n",dimSort, 'X', runningTime_GPU_overlap2, Join_Total_Time_GPU);
    hipFree(doXMBR);
    hipFree(dbXMBR);
    hipFree(djxyCounter);
//------------------------------------------------------------------------------------------------------
return(0);

/*
//---------------------------------- CMF filter for Polygon Test operation -----------------------------
    StartTimer(&start_GPU, &stop_GPU);
    int *djxy2IndexList, *djPiPIndexList, *dPiPFlag, *djoinFlag;
    char* dPiPType;
    long eiNum, pairNum3, pipNum, workLoadNum;
    coord_t *dcMBR, *dbMBR2, *doMBR2;
    CopyToGPU((void**)&doMBR2, seqOMBR2, sizeof(coord_t)*oPolNum*4, "doMBR2", 1);
    CopyToGPU((void**)&dbMBR2, seqMBR2, sizeof(coord_t)*bPolNum*4, "dbMBR2", 1);

    GetCMBR(pairNum, djxyVector, dbMBR2, doMBR2, &dcMBR, &djPiPIndexList, &dPiPFlag, &dPiPType, &djoinFlag, &pipNum);

    float runningTime_GPU_PiPCMF;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_PiPCMF);
    printf("\nGPU Running Time for CMF Filter for Point in Polygon Test: %f (%f)\n", runningTime_GPU_PiPCMF, Join_Total_Time_GPU);
//------------------------------------------------------------------------------------------------------

//---------------------------------- Point in Polygon Test operation -----------------------------------
    StartTimer(&start_GPU, &stop_GPU);
    long wNum;

    wNum=PointInPolygonTest(bCoords, oCoords, pairNum, pipNum, djxyVector, djPiPIndexList, dPiPType, dbVPSNum, doVPSNum, dPiPFlag, djoinFlag);
    
    printf("\n\tNumber of within pairs: %ld\n", wNum);

    float runningTime_GPU_PiP;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_PiP);
    printf("\nGPU Running Time for Point in Polygon Test: %f (%f)\n", runningTime_GPU_PiP, Join_Total_Time_GPU);
 
//------------------------------------------------------------------------------------------------------

//--------------------------- Applying Common MBR Filtering (novel approach) ---------------------------
    StartTimer(&start_GPU, &stop_GPU);
    poly_size_t *dbEdgeList, *doEdgeList;
    long *dbEdgePSCounter, *doEdgePSCounter, *dWorkLoadPSCounter;


    CountCMF(bCoords, oCoords, pairNum, djxyVector, djoinFlag, dbVNum, doVNum, dbVPSNum, doVPSNum, dcMBR, &dbEdgePSCounter, &doEdgePSCounter, &dWorkLoadPSCounter, &djxy2IndexList, &dbEdgeList, &doEdgeList, &eiNum, &workLoadNum);

    float runningTime_GPU_CCMF;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_CCMF);
    printf("\nGPU Running Time for Counting Common MBR Filter: %f (%f)\n", runningTime_GPU_CCMF, Join_Total_Time_GPU);

    StartTimer(&start_GPU, &stop_GPU);

    ApplyCMF(bCoords, oCoords, pairNum, djxyVector, eiNum, djxy2IndexList, dbVNum, doVNum, dbVPSNum, doVPSNum, dcMBR, dbEdgePSCounter, doEdgePSCounter, dbEdgeList, doEdgeList);

    float runningTime_GPU_ACMF;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_ACMF);
    printf("\nGPU Running Time for Applying Common MBR Filter: %f (%f)\n", runningTime_GPU_ACMF, Join_Total_Time_GPU);
    hipFree(dcMBR);
//------------------------------------------------------------------------------

//--------------------------- Join/Overlay operations --------------------------
    StartTimer(&start_GPU, &stop_GPU);
    int* dSegmentIntersectJoinFlag;
    pairNum3=SegmentIntersectJoin(bCoords, oCoords, eiNum, djxyVector, djxy2IndexList, dbVPSNum, doVPSNum, dbEdgePSCounter, doEdgePSCounter, dbEdgeList, doEdgeList, &dSegmentIntersectJoinFlag);

    printf("\n\tActual number of intersected polygon pairs: %ld\n", pairNum3);
    float runningTime_GPU_CEI;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_CEI);
    printf("\nGPU Running Time for Counting Edge Intersecions: %f (%f)\n", runningTime_GPU_CEI, Join_Total_Time_GPU);
//------------------------------------------------------------------------------
*/


    hipDeviceReset();
    return 0;
}
