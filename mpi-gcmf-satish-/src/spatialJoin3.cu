#include <stdio.h>
/*#include "GPU_Manage.h"
#include "Types.h"
#include "Constants.h"
#include "GPU_Test.h"
#include "GPU_MBR.h"
#include "IO.h"
#include "SEQ_Overlay.h"
#include "GPU_Utility.h"
#include "Data_Visualization.h"
#include "Join.h"*/
#include "ST_Intersect.h"

hipEvent_t start_GPU, stop_GPU;

int main(int argc, char* argv[]){  
    float Join_Total_Time_SEQ=0, Join_Total_Time_GPU=0;
    hipError_t cudaMemError;
//------------------------ Console Input ---------------------------------- 
/*
First user input: dimSort
	1: Sorting just based on one dimension (default is X)
	0: Sorting based on both X and Y dimensions
Second user input: dimSelect
	If dimSort=1, this argument define which dimension should be picked for sorting (Values could be 'X' or 'Y')
*/
    int dimSort=1, dimSelect=1;
    if(argc<2){
       dimSort=1;
       dimSelect=0;
    }
    else if(argc<3){
      if(argv[1][0]=='2')dimSort=2;
      else dimSort=1;
      dimSelect=0;
    }
    else if(argc<4){
      if(argv[2][0]=='y')dimSelect=1;
      else dimSelect=0;
      if(argv[1][0]=='2'){dimSort=2;dimSelect=0;}
      else dimSort=1;
    }
//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
//=============================== SEQUENTIAL RUN ===============================
//------------------------------------------------------------------------------    
//------------------------------------------------------------------------------    
    coord_t* baseCoords=(coord_t*)malloc(MAX_POLYS_BASE*4*AVG_VERTEX_PER_BASE_POL*sizeof(coord_t));
    coord_t* overlayCoords=(coord_t*)malloc(MAX_POLYS_OVERLAY*4*AVG_VERTEX_PER_OVERLAY_POL*sizeof(coord_t));
    int *bVNum=(int*)malloc(sizeof(int)*MAX_POLYS_BASE);
    int *oVNum=(int*)malloc(sizeof(int)*MAX_POLYS_OVERLAY);
    long *bVPSNum=(long*)malloc(sizeof(long)*MAX_POLYS_BASE);
    long *oVPSNum=(long*)malloc(sizeof(long)*MAX_POLYS_OVERLAY);
    long bPolNum, oPolNum, bVNumSum=0, oVNumSum=0;    
    mbr_t* seqMBR=(mbr_t*)malloc(MAX_POLYS_BASE*4*sizeof(mbr_t));
    mbr_t* seqOMBR=(mbr_t*)malloc(MAX_POLYS_OVERLAY*4*sizeof(mbr_t));
    coord_t* seqMBR2=(coord_t*)malloc(MAX_POLYS_BASE*4*sizeof(coord_t));
    coord_t* seqOMBR2=(coord_t*)malloc(MAX_POLYS_OVERLAY*4*sizeof(coord_t));
    //=================== Reading First(base) Polygon ==========================

    char baseFileName[100], overlayFileName[100];
    switch(DATASET){
       case 1:
         strcpy(baseFileName, "../admin_states.txt");
	 strcpy(overlayFileName, "../urban_areas.txt");
         break;
       case 2:
         strcpy(baseFileName, "bases_242.txt");
         strcpy(overlayFileName, "overlay_300.txt");
         break;
       case 3:
         strcpy(baseFileName, "block_boundaries.txt");
         strcpy(overlayFileName, "water_bodies.txt");
         break;
    }
    bPolNum=ReadTextFormatPolygon(baseFileName,bVNum, bVPSNum, seqMBR, seqMBR2, baseCoords, &bVNumSum, 1, MAX_POLYS_BASE);    
    printf("\n%lu Polygons with %lu vertices in total.\n",bPolNum,bVNumSum);
    oPolNum=ReadTextFormatPolygon(overlayFileName, oVNum, oVPSNum, seqOMBR, seqOMBR2, overlayCoords, &oVNumSum, 1, MAX_POLYS_OVERLAY); 
    printf("\n%lu Polygons with %lu vertices in total.\n",oPolNum,oVNumSum);
    //==========================================================================

//=========================== Reseting GPU device ==============================
    hipError_t error_reset=hipDeviceReset();    
    if(error_reset!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error_reset) );
       exit(-1);
    }
    hipDeviceReset();
    
    //==================== Running Kernel (CreateMBR) =========================


    ST_Intersect(bPolNum, oPolNum, baseCoords, overlayCoords, bVNum, oVNum, bVPSNum, oVPSNum, seqMBR, seqOMBR, seqMBR2, seqOMBR2);

   hipDeviceReset();
   //==============================================================================
return 0;
}
