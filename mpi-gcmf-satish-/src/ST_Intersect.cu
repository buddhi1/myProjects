
#include "stdio.h"
#include "GPU_Manage.h"
#include "Types.h"
#include "Constants.h"
#include "GPU_Test.h"
#include "GPU_MBR.h"
#include "IO.h"
#include "SEQ_Overlay.h"
#include "GPU_Utility.h"
#include "Data_Visualization.h"
#include "Join.h"

// int main() {
//   return 0;
// }

int ST_Intersect(long bPolNum, long oPolNum, coord_t* baseCoords, coord_t* overlayCoords, int* bVNum, int* oVNum, long *bVPSNum, long *oVPSNum, 
                 mbr_t* seqMBR, mbr_t* seqOMBR, coord_t *seqMBR2, coord_t* seqOMBR2, int * jPairs){

    printf("In ST_Intersect \n");
    fflush(stdout);
    
    long bVNumSum=bVPSNum[bPolNum-1], oVNumSum=oVPSNum[oPolNum-1];
    float Join_Total_Time_SEQ=0, Join_Total_Time_GPU=0;
    hipError_t cudaMemError;
    int dimSort=1, dimSelect=0, retVal=0;

//=============================== SEQUENTIAL RUN ===============================
//------------------------------------------------------------------------------    

//PrintPolygon(baseCoords+2*bVPSNum[1485], bVNum[1486]);
//printf("\n\n\n");
//PrintPolygon(overlayCoords+2*oVPSNum[10], oVNum[11]);
//return;

//=========================== Reseting GPU device ==============================
    //====================== Transfering data to GPU ==========================
    hipStream_t gStream;
    hipError_t stEr=hipStreamCreate(&gStream);
    if(stEr!=hipSuccess){printf("\nError in creating stream!\n");return(0);}
    hipEvent_t start_GPU, stop_GPU;
    StartTimer(&start_GPU, &stop_GPU);
   
    int *dbVNum, *doVNum;
    coord_t *oCoords, *bCoords;
    mbr_t *doMBR, *dbMBR;
    long *dbVPSNum, *doVPSNum;

    //----------- Transfering polygon number variables to GPU ---------------
    CopyToGPU((void**)&dbVNum, bVNum, sizeof(int)*bPolNum, "dbVNum", 1);
    CopyToGPU((void**)&doVNum, oVNum, sizeof(int)*oPolNum, "doVNum", 1);
    CopyToGPU((void**)&dbVPSNum, bVPSNum, sizeof(long)*bPolNum, "dbVPSNum", 1);
    CopyToGPU((void**)&doVPSNum, oVPSNum, sizeof(long)*oPolNum, "doVPSNum", 1);
    //-----------------------------------------------------------------------
    //------------- Transfering polygon coordinates to GPU i-----------------
    CopyToGPU((void**)&bCoords, baseCoords, sizeof(coord_t)*2*bVNumSum, "bCoords", 1);
    CopyToGPU((void**)&oCoords, overlayCoords, sizeof(coord_t)*2*oVNumSum, "oCoords", 1);
    //-----------------------------------------------------------------------
    //----------------------- Transfering MBRs to GPU -----------------------
    CopyToGPU((void**)&dbMBR, seqMBR, 4*sizeof(mbr_t)*bPolNum, "dbMBR", 1);
    CopyToGPU((void**)&doMBR, seqOMBR, 4*sizeof(mbr_t)*oPolNum, "doMBR", 1);
    //-----------------------------------------------------------------------
    GPUSync("Transfering data to GPU");

    if(DEBUG_MODE){
      float runningTime_GPU_TransferData;
      Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_TransferData);
      printf("\n\nGPU running time for transfering data to GPU: %f (%f)\n",runningTime_GPU_TransferData, Join_Total_Time_GPU);
    }
//==============================================================================


//--------------------------- Find Overlaping MBRs (novel approach) ---------------------------
    StartTimer(&start_GPU, &stop_GPU);

    int *djxyCounter, *djxyVector, polNum=bPolNum+oPolNum; 
    cudaMemError=hipMalloc((void**)&djxyCounter,sizeof(int)*(polNum));

    long pairNum=SortBaseMBROverlap(gStream, bPolNum, oPolNum, dbMBR, doMBR, &djxyCounter, &djxyVector, dimSort, dimSelect);
    float runningTime_GPU_overlap2;
   
    if(DEBUG_MODE){
      printf("\n\n\tPolygon pairs candidate: %ld\n", pairNum);
      Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_overlap2);
      printf("\nGPU Running Time For Computing MBR intersection (new approach %dD [dim:%c] ): %f (%f)\n",dimSort, 'X', runningTime_GPU_overlap2, Join_Total_Time_GPU);
    }
    hipFree(doMBR);
    hipFree(dbMBR);
    hipFree(djxyCounter);
    if(pairNum==0)return(0);
//------------------------------------------------------------------------------

//--------------------------- CMF filter for Polygon Test operation --------------------------
    StartTimer(&start_GPU, &stop_GPU);
    int *djxy2IndexList, *djPiPIndexList;
    char* dPiPType, *dPiPFlag, *djoinFlag;
    long eiNum, pairNum3, *pipNum, workLoadNum;
    coord_t *dcMBR, *dbMBR2, *doMBR2;
    CopyToGPU((void**)&doMBR2, seqOMBR2, sizeof(coord_t)*oPolNum*4, "doMBR2", 1);
    CopyToGPU((void**)&dbMBR2, seqMBR2, sizeof(coord_t)*bPolNum*4, "dbMBR2", 1);
    GPUSync("Transfering data to GPU");

    GetCMBR(gStream, pairNum, djxyVector, dbMBR2, doMBR2, &dcMBR, &djPiPIndexList, &dPiPFlag, &dPiPType, &djoinFlag, &pipNum);
 
    if(DEBUG_MODE){
       float runningTime_GPU_PiPCMF;
       Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_PiPCMF);
       printf("\nGPU Running Time for CMF Filter for Point in Polygon Test: %f (%f)\n", runningTime_GPU_PiPCMF, Join_Total_Time_GPU);
    }
//------------------------------------------------------------------------------
//--------------------------- Point in Polygon Test operation --------------------------
    StartTimer(&start_GPU, &stop_GPU);
    long wNum;


    wNum=PointInPolygonTest(bCoords, oCoords, pairNum, *pipNum, djxyVector, djPiPIndexList, dPiPType, dbVPSNum, doVPSNum, dPiPFlag, djoinFlag);
    
    if(DEBUG_MODE)printf("\n\tNumber of within pairs: %ld\n", wNum);
    retVal+=wNum;

    //PrintPairs(djxyVector, dPiPFlag, pairNum);
//GPUPrintVector(pairNum2, dEdgeIntersectCounter, 1);

    if(DEBUG_MODE){
      float runningTime_GPU_PiP;
      Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_PiP);
      printf("\nGPU Running Time for Point in Polygon Test: %f (%f)\n", runningTime_GPU_PiP, Join_Total_Time_GPU);
    }
 
//------------------------------------------------------------------------------


//--------------------------- Applying Common MBR Filtering (novel approach) ---------------------------
    StartTimer(&start_GPU, &stop_GPU);
    poly_size_t *dbEdgeList, *doEdgeList;
    long *dbEdgePSCounter, *doEdgePSCounter, *dWorkLoadPSCounter;


     CountCMF(gStream, bCoords, oCoords, pairNum, djxyVector, djoinFlag, dbVNum, doVNum, dbVPSNum, doVPSNum, dcMBR, &dbEdgePSCounter, &doEdgePSCounter, &dWorkLoadPSCounter, &djxy2IndexList, &dbEdgeList, &doEdgeList, &eiNum, &workLoadNum);

    //printf("\n\tPolygon pair candidates after Applying CMF filter: %ld\n", eiNum);
    if(DEBUG_MODE){
      float runningTime_GPU_CCMF;
      Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_CCMF);
      printf("\nGPU Running Time for Counting Common MBR Filter: %f (%f)\n", runningTime_GPU_CCMF, Join_Total_Time_GPU);
    }


    StartTimer(&start_GPU, &stop_GPU);

    ApplyCMF(bCoords, oCoords, pairNum, djxyVector, eiNum, djxy2IndexList, dbVNum, doVNum, dbVPSNum, doVPSNum, dcMBR, dbEdgePSCounter, doEdgePSCounter, dbEdgeList, doEdgeList);
    //GPUPrintVector(pairNum*2, djxyVector, 0);
    //GPUPrefixsumTest(dbEdgeCounter, dbEdgePSCounter, pairNum, 1);
    //GPUPrefixsumTest(doEdgeCounter, doEdgePSCounter, pairNum, 1);

    if(DEBUG_MODE){
      float runningTime_GPU_ACMF;
      Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_ACMF);
      printf("\nGPU Running Time for Applying Common MBR Filter: %f (%f)\n", runningTime_GPU_ACMF, Join_Total_Time_GPU);
    }
    hipFree(dcMBR);

//------------------------------------------------------------------------------

//--------------------------- Join/Overlay operations --------------------------
    StartTimer(&start_GPU, &stop_GPU);
    char* dSegmentIntersectJoinFlag;
     pairNum3=SegmentIntersectJoin(bCoords, oCoords, eiNum, djxyVector, djxy2IndexList, dbVPSNum, doVPSNum, dbEdgePSCounter, doEdgePSCounter, dbEdgeList, doEdgeList, &dSegmentIntersectJoinFlag);
    //pairNum3=SegmentIntersectJoin2(bCoords, oCoords, eiNum, djxyVector, djxy2IndexList, dbVPSNum, doVPSNum, dbEdgePSCounter, doEdgePSCounter, dWorkLoadPSCounter, workLoadNum, dbEdgeList, doEdgeList, &dSegmentIntersectJoinFlag);
    //PrintPairs(djxyVector, dPiPFlag, pairNum);

    retVal+=pairNum3;

    if(DEBUG_MODE){
      printf("\n\tActual number of intersected polygon pairs: %ld\n", pairNum3);
      float runningTime_GPU_CEI;
      Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_CEI);
      printf("\nGPU Running Time for Counting Edge Intersecions: %f (%f)\n", runningTime_GPU_CEI, Join_Total_Time_GPU);
    }
//------------------------------------------------------------------------------

    GPUSync("SegmentIntersect");


   char *pipFlag, *joinFlag, *jxyVector, *jxyIndexList, mappedIndx; 
   CopyFromGPU((void**)&pipFlag, dPiPFlag, sizeof(char)*pairNum, 1);
   CopyFromGPU((void**)&joinFlag, dSegmentIntersectJoinFlag, sizeof(char)*eiNum, 1);
   CopyFromGPU((void**)&jxyIndexList, djxy2IndexList, sizeof(int)*eiNum, 1);
   CopyFromGPU((void**)&jxyVector, djxyVector, 2*sizeof(int)*pairNum, 1);
   jPairs=(int*)malloc(sizeof(int)*2*retVal);

   int indx=0;
   for(int i=0;i<pairNum;i++){
      if(pipFlag[i]==1){
         jPairs[indx*2]=jxyVector[2*i];
         jPairs[indx*2+1]=jxyVector[2*i+1];
         indx++;
     }
      else if(i<eiNum){
         if(joinFlag[i]==1){
            mappedIndx=jxyIndexList[i];
            jPairs[indx*2]=jxyVector[2*mappedIndx];
            jPairs[indx*2+1]=jxyVector[2*mappedIndx+1];
            indx++;
         }
      }
      if(indx>retVal){
        printf("\nToo many outputs!\n");
        break;
      }
   }

   hipFree(dPiPFlag);
   hipFree(djxyVector);
   hipFree(djoinFlag);
   free(pipFlag);
   free(joinFlag);
   free(jxyVector);

   hipDeviceReset();
   return(retVal);

//==============================================================================
}
