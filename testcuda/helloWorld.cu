
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void test_kernel(int *a, int *b, int *c, int device_id) {
        int tid = blockIdx.x*blockDim.x + threadIdx.x;
        // printf("here");
        if(tid < N) {
                c[tid] = a[tid] + b[tid];
                // printf("c: %d ", c[tid]);
        }
        // printf("Hello From GPU device id %u %d\n", device_id, threadIdx.x);
}

int main() {
        // int nDevices;

        // cudaGetDeviceCount(&nDevices);
        // for (int i = 0; i < nDevices; i++) {
        //         cudaDeviceProp prop;
        //         cudaGetDeviceProperties(&prop, i);
        //         printf("Device Number: %d\n", i);
        //         printf("  Device name: %s\n", prop.name);
        //         printf("  Memory Clock Rate (KHz): %d\n",
        //         prop.memoryClockRate);
        //         printf("  Memory Bus Width (bits): %d\n",
        //         prop.memoryBusWidth);
        //         printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
        //         2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        // }

        int devices_count;
        int a[N], b[N], c[N];
        int *dev_a, *dev_b, *dev_c;
        hipGetDeviceCount(&devices_count);

        printf("count %d\n", devices_count);

        // populate data
        for(int i=0; i<N; ++i){
                a[i]=i;
                b[i]=i;
        } 

        for (unsigned int device_id = 0; device_id < 2; device_id++)
        {
                hipSetDevice (device_id);
                // const unsigned int chunk_size = chunk_ends[device_id] - chunk_begins[device_id];
                // const unsigned char *host_chunk_input = img->data.get () + chunk_begins[device_id];
                // unsigned char *host_chunk_output = host_result + chunk_begins[device_id];
                // cudaMemcpy (devices_inputs[device_id], host_chunk_input, chunk_size * sizeof (unsigned char), cudaMemcpyHostToDevice);
                
                // gpu_div_kernel_vec<div> <<<block_sizes[device_id], threads_per_block>>> (devices_inputs[device_id], devices_outputs[device_id]);
                
                // cudaMemcpy (host_chunk_output, devices_outputs[device_id], chunk_size * sizeof (unsigned char), cudaMemcpyDeviceToHost);

                hipMalloc((void**)&dev_a, sizeof(int)*N);
                hipMalloc((void**)&dev_b, sizeof(int)*N);
                hipMalloc((void**)&dev_c, sizeof(int)*N);
                hipMemcpy(dev_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
                hipMemcpy(dev_b, b, sizeof(int)*N, hipMemcpyHostToDevice);
                test_kernel<<<1, N>>>(dev_a, dev_b, dev_c, device_id);
                hipMemcpy(dev_c, c, sizeof(int)*N, hipMemcpyDeviceToHost);

                // for(int i=0; i<N; ++i) {
                //         printf("%d ", c[i]);
                // }
                // printf("\n");
        }

    return 0;
}