/* multiply.cu */
#include <hip/hip_runtime.h>
// #include <cuda_runtime.h>
#include <stdio.h>
 
 __global__ void __multiply__ ()
 {
     printf("Hello World from GPU!!!\n");
 }
 
 extern "C" void call_me_maybe()
{
     /* ... Load CPU data into GPU buffers  */
 
     __multiply__ <<<1,2>>> ();
     hipDeviceSynchronize();
     printf("From host cuda\n");
 
     /* ... Transfer data from GPU to CPU */
}